#include <iostream>
#include <hip/hip_runtime.h>
#include <cusparseLt.h>

static const char* status_str(hipsparseStatus_t s){
  switch(s){
    case HIPSPARSE_STATUS_SUCCESS: return "SUCCESS";
    case HIPSPARSE_STATUS_NOT_INITIALIZED: return "NOT_INITIALIZED";
    case HIPSPARSE_STATUS_ALLOC_FAILED: return "ALLOC_FAILED";
    case HIPSPARSE_STATUS_INVALID_VALUE: return "INVALID_VALUE";
    case HIPSPARSE_STATUS_ARCH_MISMATCH: return "ARCH_MISMATCH";
    case HIPSPARSE_STATUS_MAPPING_ERROR: return "MAPPING_ERROR";
    case HIPSPARSE_STATUS_EXECUTION_FAILED: return "EXECUTION_FAILED";
    case HIPSPARSE_STATUS_INTERNAL_ERROR: return "INTERNAL_ERROR";
    case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED: return "MATRIX_TYPE_NOT_SUPPORTED";
    case HIPSPARSE_STATUS_ZERO_PIVOT: return "ZERO_PIVOT";
    case HIPSPARSE_STATUS_NOT_SUPPORTED: return "NOT_SUPPORTED";
    case HIPSPARSE_STATUS_INSUFFICIENT_RESOURCES: return "INSUFFICIENT_RESOURCES";
    default: return "UNKNOWN";
  }
}

int main() {
    int ndev = 0;
    hipGetDeviceCount(&ndev);
    if (ndev <= 0) { std::cerr << "No CUDA devices visible\n"; return 2; }
    hipDeviceProp_t p{};
    hipGetDeviceProperties(&p, 0);
    std::cout << "GPU: " << p.name << "  CC " << p.major << "." << p.minor << "\n";

    cusparseLtHandle_t handle;
    hipsparseStatus_t s = cusparseLtInit(&handle);
    if (s != HIPSPARSE_STATUS_SUCCESS) {
        std::cerr << "cusparselt FAILED: " << (int)s << " (" << status_str(s) << ")\n";
        if (s == HIPSPARSE_STATUS_ARCH_MISMATCH) {
            std::cout << "cusparselt FAILED: ARCH_MISMATCH - wrong binary for this GPU\n";
            return 1; // fail on ARCH_MISMATCH to indicate compatibility issue
        }
        return 1;
    }
    std::cout << "cusparselt OK\n";
    cusparseLtDestroy(&handle);
    return 0;
}
